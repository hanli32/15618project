#include "hip/hip_runtime.h"
#include <cusp/csr_matrix.h>
#include <cusp/monitor.h>
#include <cusp/gallery/poisson.h>
#include <cusp/krylov/cg.h>
#include <iostream>
#include "CycleTimer.h"
#include "serial_cg.h"

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

/**
 * check correctness of solver solution
 * @param  x 
 * @param  b 
 * @return err
 */
float check_err(float *x, float *b, int N, int* I, int* J, float* val) {
    float rsum, diff, err = 0.0;
    for (int i = 0; i < N; i++) {
        rsum = 0.0;

        for (int j = I[i]; j < I[i + 1]; j++) {
            rsum += val[j] * x[J[j]];
        }

        diff = fabs(rsum - b[i]);

        if (diff > err) {
            err = diff;
        }
    }
    return err;
}

/**
 * generate a random tridiagonal symmetric matrix
 * @param I   offset
 * @param J   column 
 * @param val values
 * @param N   matrix row/col size
 * @param nz  nonzero size
 */
void genTridiag(int *I, int *J, float *val, int N, int nz)
{
    I[0] = 0, J[0] = 0, J[1] = 1;
    val[0] = (float)rand()/RAND_MAX + 10.0f;
    val[1] = (float)rand()/RAND_MAX;
    int start;

    for (int i = 1; i < N; i++)
    {
        if (i > 1)
        {
            I[i] = I[i-1]+3;
        }
        else
        {
            I[1] = 2;
        }

        start = (i-1)*3 + 2;
        J[start] = i - 1;
        J[start+1] = i;

        if (i < N-1)
        {
            J[start+2] = i + 1;
        }

        val[start] = val[start-1];
        val[start+1] = (float)rand()/RAND_MAX + 10.0f;

        if (i < N-1)
        {
            val[start+2] = (float)rand()/RAND_MAX;
        }
    }

    I[N] = nz;
}
/**
 * cuSPARSE version cg solver
 * @return overall duration
 */
double cusparse_cg(int N, int nz, int* I, int* J, float* val) 
{
    float *x;
    float *rhs;
    float a, b, na, r0, r1;
    int *d_col, *d_row;
    float *d_val, *d_x, dot;
    float *d_r, *d_p, *d_Ax;
    int k;
    float alpha, beta, alpham1;

    x = (float *) malloc(sizeof(float) * N);
    rhs = (float *) malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
        rhs[i] = 1.0;
        x[i] = 0.0;
    }

    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);

    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);

    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);

    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    (hipMalloc((void **) &d_col, nz * sizeof(int)));
    (hipMalloc((void **) &d_row, (N + 1) * sizeof(int)));
    (hipMalloc((void **) &d_val, nz * sizeof(float)));
    (hipMalloc((void **) &d_x, N * sizeof(float)));
    (hipMalloc((void **) &d_r, N * sizeof(float)));
    (hipMalloc((void **) &d_p, N * sizeof(float)));
    (hipMalloc((void **) &d_Ax, N * sizeof(float)));

    hipMemcpy(d_col, J, nz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, I, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, nz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_r, rhs, N * sizeof(float), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.;

    hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x,
                   &beta, d_Ax);

    hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);
    hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);

    k = 1;

    while (r1 > TOL * TOL && k <= MAX_ITER) {
        if (k > 1) {
            b = r1 / r0;
            hipblasStatus_t = hipblasSscal(cublasHandle, N, &b, d_p, 1);
            hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);
        }
        else {
            hipblasStatus_t = hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);
        }

        hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col,
                       d_p, &beta, d_Ax);
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);
        a = r1 / dot;

        hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);
        na = -a;
        hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);

        r0 = r1;
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
        hipDeviceSynchronize();
        printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }

    double time_cost = CycleTimer::currentSeconds() - startTime;

    hipMemcpy(x, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
    // float err = check_err(x, rhs);

    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    free(x);
    free(rhs);
    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ax);

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

    // printf("Test Summary:  Error amount = %f\n", err);
    return time_cost;
}

/**
 * CUSP version cg solver
 * @return overall duration
 */
template <class LinearOperator>
double cusp_cg(LinearOperator& A) {

    cusp::csr_matrix<int,float,cusp::device_memory> d_A(A);
    // allocate storage for solution (x) and right hand side (b)
    cusp::array1d<float, cusp::device_memory> x(d_A.num_rows, 0);
    cusp::array1d<float, cusp::device_memory> b(d_A.num_rows, 1);

    // set stopping criteria:
    //  iteration_limit    = 100
    //  relative_tolerance = TOL * TOL
    //  absolute_tolerance = 0
    //  verbose            = true
    cusp::monitor<float> monitor(b, 100, TOL * TOL, 0, true);

    // set preconditioner (identity)
    cusp::identity_operator<float, cusp::device_memory> M(d_A.num_rows, d_A.num_rows);

    double startTime = CycleTimer::currentSeconds();
    // solve the linear system A * x = b with the Conjugate Gradient method
    cusp::krylov::cg(d_A, x, b, monitor, M);

    double time_cost = CycleTimer::currentSeconds() - startTime;

    return time_cost;
}

int main(int argc, char **argv)
{
    // Check the number of parameters
    if (argc < 2) {
        // Tell the user how to run the program
        std::cerr << "Usage: N " << std::endl;
        return 1;
    }

    int M = 0, N = 0, nz = 0, *I, *J;
    float *val; 

    /* Generate a random tridiagonal symmetric matrix in CSR format */
    M = N = std::atoi(argv[1]);//1048576;
    std::cout<<"============== N = "<< N <<" ====================\n";
    nz = (N-2)*3 + 4;
    I = (int *)malloc(sizeof(int)*(N+1));
    J = (int *)malloc(sizeof(int)*nz);
    val = (float *)malloc(sizeof(float)*nz);
    double* val_double = (double *)malloc(sizeof(double)*nz);
    genTridiag(I, J, val, N, nz);

    /* Generate CUSP matrix data type */
    cusp::csr_matrix<int, float, cusp::host_memory> A(M, N, nz);
    for (int i = 0; i < N + 1; i++) {
        A.row_offsets[i] = I[i];
    }
    for (int i = 0; i < nz; i++) {
        A.column_indices[i] = J[i];
        A.values[i] = val[i];
        val_double[i] = val[i];
    }

    double cusparse_time = cusparse_cg(N, nz, I, J, val);
    double cusp_time = cusp_cg(A);
    double serial_time = serial_cg(I, J, val_double, N);

    for (int i = 0; i < 5; i++) {
        cusparse_time = std::min(cusparse_time, cusparse_cg(N, nz, I, J, val));
        cusp_time = std::min(cusp_time, cusp_cg(A));
        serial_time = std::min(serial_time, serial_cg(I, J, val_double, N));
    }

    free(I);
    free(J);
    free(val);
    free(val_double);
    printf("cusparse_time: %f\ncusp_time: %f\nserial_time: %f\n", 
        cusparse_time * 1000, cusp_time * 1000, serial_time * 1000);
}
